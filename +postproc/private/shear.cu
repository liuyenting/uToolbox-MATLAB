#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

#include <cstdint>              // uint16_t, uint8_t
#include <cassert>              // assert
#include <algorithm>            // std::fill
#include <hip/hip_runtime_api.h>   // cuda*

#include "util/structparser.hpp"    // parseFloatField
#include "util/gpuerror.cuh"        // gpuCheckError
#include "kernels/shear.cuh"        // shearLayer

#include "shear.cuh"

Shear::Shear() {
    // Reset the acquisition paramters.
    acqParams = {};

    // Reset copy related parameters.
    cpParam = {};

    // Reset the layer counter only, since this will trigger the copy.
    nLayers = 0;

    // Initialize MathWorks GPU API.
    mxInitGPU();
}

void Shear::setAcqParam(const mxArray *in) {
    acqParams.objAngle = parseFloatField(in, "ObjectiveAngle");
    acqParams.zStepWidth = parseFloatField(in, "ZStepWidth");
    acqParams.pixelWidth = parseFloatField(in, "PixelWidth");

    mexPrintf(" ** Acquisition paramter parsed...\n    - Angle: %f[deg]\n    - Z Width: %f[rad]\n    - Px Size: %f[rad]\n",
              acqParams.objAngle, acqParams.zStepWidth, acqParams.pixelWidth);

    // Force reset in order to trigger reallocate the workspace.
    nLayers = 0;
    // TODO: Free the WS if hipArray is not NULL.
}

void Shear::loadStack(const mxArray *in) {
    // Turn mxArray to mxGPUArray.
    const mxGPUArray *inArr = mxGPUCreateFromMxArray(in);

    if (nLayers == 0) {
        const size_t *inSize = (const size_t *)mxGPUGetDimensions(inArr);
        preallocateWorkspace(inSize);
        setupImageCopyParameter();
    }

    const uint16_t *inImg = (const uint16_t *)mxGPUGetDataReadOnly(inArr);
    copyImageToDevice(inImg);

    mxGPUDestroyGPUArray(inArr);
}

void Shear::preallocateWorkspace(const size_t *_oriSize) {
    // Pre-calculate the image sizes along the pipeline.
    saveOldSize(_oriSize);
    estimateNewSize();

    mexPrintf(" ** Input size [%d, %d, %d] ... Output size [%d, %d, %d]\n",
              oldSize[0], oldSize[1], nLayers,
              newSize[0], newSize[1], nLayers);

    generateWorkspace();
}

void Shear::saveOldSize(const size_t *size) {
    std::copy(size, size+2, oldSize);
    nLayers = size[2];
}

void Shear::estimateNewSize() {
    // Alias for the original dimension.
    size_t nx = oldSize[0];
    size_t ny = oldSize[1];

    // Calculate amount of shifted pixels.
    size_t diff = static_cast<size_t>(acqParams.shearFactor()*(ny-1));

    // Fill the values.
    newSize[0] = nx + diff;
    newSize[1] = ny;
}

/*
void Shear::estimateRotatedSize() {
    // Check whether we have to bypass rotating.
    if (acqParams.noRotate) {
        std::copy(shSize, shSize+2, rotSize);
        return;
    }

    // Alias for the sheared size.
    size_t nx = shSize[0];
    size_t ny = shSize[1];

    // Unit length.
    float unitSize[4][2] = { {1, 1}, {-1, 1}, {-1, -1}, {1, -1} };
    // Generate reverse matrix.
    fillRotationMatrix(-acqParams.stackRotatedAngle());
    // Apply rotation to all the end points.
    for (uint8_t i = 0; i < 4; i++) {
        float x = unitSize[i][0] - 1.0f;
        float y = unitSize[i][1] - 1.0f;

        float u = rotMat[0]*x + rotMat[1]*y;
        float v = rotMat[2]*x + rotMat[3]*y;

        unitSize[i][0] = u;
        unitSize[i][1] = v;
    }
    // Find the minimum and maximum value of each axis.
    float uMin, uMax, vMin, vMax;
    uMin = uMax = unitSize[0][0];
    vMin = vMax = unitSize[0][1];
    for (uint8_t i = 1; i < 4; i++) {
        if (unitSize[i][0] < uMin) {
            uMin = unitSize[i][0];
        } else if (unitSize[i][0] > uMax) {
            uMax = unitSize[i][0];
        }

        if (unitSize[i][1] < vMin) {
            vMin = unitSize[i][1];
        } else if (unitSize[i][1] > vMax) {
            vMax = unitSize[i][1];
        }
    }
    // Calculate the length.
    float nu = ceil(uMax - uMin);
    assert(nu > 0 );
    float nv = ceil(vMax - vMin);
    assert(nv > 0);

    // Fill back to the array.
    rotSize[0] = (size_t)(nu * nx/2.0f);
    rotSize[1] = (size_t)(nv * ny/2.0f);

    // TODO: Calcualte the cropping factor.

    mexWarnMsgIdAndTxt(CORE_MSGID,
                       " >> Rotated size: [%d, %d, %d]",
                       rotSize[0], rotSize[1], nLayers);
}
*/

void Shear::generateWorkspace() {
    hipMemcpyToSymbol(HIP_SYMBOL(devNewSize), newSize, sizeof(size_t)*2,
                       0, hipMemcpyHostToDevice);
    gpuCheckError("Failed to copy workspace sizes to device constant memory.");

    const float shFact = acqParams.shearFactor();
    hipMemcpyToSymbol(HIP_SYMBOL(devShFact), &shFact, sizeof(float),
                       0, hipMemcpyHostToDevice);

    // NOTE: hipChannelFormatDesc cannot reuse, otherwise it will cause memory
    // deallocation error.
    hipChannelFormatDesc oldChDesc =
        hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipMalloc3DArray(&devOldImg,
                      &oldChDesc,
                      make_hipExtent(oldSize[0], oldSize[1], nLayers),
                      hipArrayLayered);
    gpuCheckError("Failed to allocate old image buffere on device.");

    const size_t nElem = newSize[0]*newSize[1]*nLayers;
    hipMalloc((void **)&devNewImg, sizeof(float)*nElem);
    gpuCheckError("Failed to allocate new image buffer on device.");
}

/*
void Shear::fillRotationMatrix(const float angle) {
    float sinVal = std::sinf(angle);
    float cosVal = std::cosf(angle);

    rotMat[0] = cosVal;
    rotMat[1] = -sinVal;
    rotMat[2] = sinVal;
    rotMat[3] = cosVal;
}
*/

void Shear::setupImageCopyParameter() {
    // Setup non-changing parameters.
    cpParam.srcPos = make_hipPos(0, 0, 0);
    cpParam.dstPos = make_hipPos(0, 0, 0);
    cpParam.dstArray = devOldImg;
    cpParam.extent = make_hipExtent(oldSize[0], oldSize[1], nLayers);
    cpParam.kind = hipMemcpyDeviceToDevice;
}

void Shear::copyImageToDevice(const uint16_t *in) {
    // Alias for the original dimension.
    const size_t nx = oldSize[0];
    const size_t ny = oldSize[1];
    // Complete the copy parameters.
    cpParam.srcPtr =
        make_hipPitchedPtr((void *)in, sizeof(uint16_t)*nx, nx, ny);
    // Start the copy.
    hipMemcpy3D(&cpParam);
    gpuCheckError("Failed to copy original image.");
}

void Shear::execute() {
    bindImageToTexture();

    const size_t nu = newSize[0];
    const size_t nv = newSize[1];
    const size_t nw = nLayers;
    // Setup the threads, grid size is ceiled.
    const dim3 blockSize(blkSize, blkSize, 1);
    const dim3 gridSize((nu+blockSize.x-1)/blockSize.x,
                        (nv+blockSize.y-1)/blockSize.y);

    // Iterate through all the layers.
    bool reversed = acqParams.isReversed();
    for (size_t iw = 0; iw < nw; iw++) {
        shearLayer<<<gridSize, blockSize>>>(devNewImg, texObj, iw, reversed);
    }
}

void Shear::bindImageToTexture() {
    // Specify texture resource.
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = devOldImg;

    // Specify texture paramters, voxels are accessed through raw coordinates.
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    // Nearest neighbor: Point, Interpolation: Linear.
    texDesc.filterMode = hipFilterModePoint;
    // Output as normalized float [0, 1] per pixel.
    texDesc.readMode = hipReadModeNormalizedFloat;
    // Access with original coordinates.
    texDesc.normalizedCoords = false;

    // Bind the array to texture.
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
    gpuCheckError("Failed to bind device data to texture.");
}

void Shear::retrieveResult(mxArray **out) {
    const size_t outSize[3] = { newSize[0], newSize[1], nLayers };
    mxGPUArray *outArr =
        mxGPUCreateGPUArray(3, outSize, mxSINGLE_CLASS, mxREAL,
                            MX_GPU_DO_NOT_INITIALIZE);
    float *outImg = (float *)mxGPUGetData(outArr);
    // NOTE: result is assumed to match the size of outImg.
    const size_t nElem = outSize[0]*outSize[1]*outSize[2];
    hipMemcpy(outImg, devNewImg, sizeof(float)*nElem,
               hipMemcpyDeviceToDevice);
    gpuCheckError("Failed to move result from device to host.");

    // Output the result to MATLAB.
    *out = mxGPUCreateMxArrayOnGPU(outArr);
    mxGPUDestroyGPUArray(outArr);
}

Shear::~Shear() {
    releaseWorkspace();
    mexPrintf(" ** GPU resources released\n");
}

void Shear::releaseWorkspace() {
    hipFreeArray(devOldImg);
    hipFree(devNewImg);
    gpuCheckError("Failed to free the resources on device.");
}
