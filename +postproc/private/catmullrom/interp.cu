#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>   // cuda*

#include "coeff.cuh"            // w0, w1, w2, w3

#include "interp.cuh"

__device__
float cubicInterp(float x,
                  float c0, float c1, float c2, float c3) {
    float r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

__device__
float pixelCubicLookup(hipTextureObject_t texObj,
                       float x, float y, float layer) {
    #define texlookup(x, y) tex2DLayered<float>(texObj, x, y, layer)

    // Integer pixel location.
    float ix = floor(x);
    float iy = floor(y);
    // Fraction for interpolation.
    float fx = x - ix;
    float fy = y - iy;

    // Perform two cubic interpolations.
    return cubicInterp(fy,
                       cubicInterp(fx, texlookup(ix-1, iy-1), texlookup(ix, iy-1), texlookup(ix+1, iy-1), texlookup(ix+2, iy-1)),
                       cubicInterp(fx, texlookup(ix  , iy  ), texlookup(ix, iy  ), texlookup(ix+1, iy  ), texlookup(ix+2, iy  )),
                       cubicInterp(fx, texlookup(ix-1, iy+1), texlookup(ix, iy+1), texlookup(ix+1, iy+1), texlookup(ix+2, iy+1)),
                       cubicInterp(fx, texlookup(ix-1, iy+2), texlookup(ix, iy+2), texlookup(ix+1, iy+2), texlookup(ix+2, iy+2)));
}
